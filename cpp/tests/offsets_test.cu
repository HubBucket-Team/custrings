
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include <numeric>
#include "../include/NVStrings.h"
#include "../include/NVCategory.h"
#include "../include/ipc_transfer.h"


//
// cd ../build
// nvcc -g -G -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/offsets_test.cu -L. -lNVStrings -lNVCategory -o offsets_test --linker-options -rpath,.:
//

int category_test( std::string& mode )
{
    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        FILE* fh = fopen("/tmp/cputx.bin","rb");

        size_t count;
        size_t offsets_size;
        size_t strs_size;
        
        fread(&count,1,sizeof(size_t),fh);

        fread(&offsets_size,1,sizeof(size_t),fh);

        int* offsets_ptr = (int*) malloc(offsets_size);
        fread(offsets_ptr,1,offsets_size,fh);

        fread(&strs_size,1,sizeof(size_t),fh);

        char* strs_ptr = (char*) malloc(strs_size);
        fread(strs_ptr,1,strs_size,fh);

        fclose(fh);

        cat = NVCategory::create_from_offsets(strs_ptr, count, offsets_ptr);

        NVStrings* strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrs[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrs,5);
        cat = NVCategory::create_from_strings(*strs);

        strs = cat->get_keys();
        strs->print();

        FILE* fh = fopen("/tmp/cputx.bin","wb");

        size_t count = strs->size();
        fwrite((void*)&count,1,sizeof(size_t),fh);

        size_t offsets_size = (count+1)*sizeof(int);
        int* offsets_ptr = (int*) malloc(offsets_size);

        int* lengths = (int*) malloc(count*sizeof(int));
        strs->byte_count(lengths, false);

        size_t strs_size = std::accumulate(lengths, lengths+count, 0);
        char* strs_ptr = (char*) malloc(strs_size);
        strs->create_offsets( strs_ptr, offsets_ptr, nullptr, false);

        fwrite((void*)&offsets_size,1,sizeof(size_t),fh);
        fwrite(offsets_ptr,1,offsets_size,fh);

        fwrite((void*)&strs_size,1,sizeof(size_t),fh);
        fwrite(strs_ptr,1,strs_size,fh);

        fclose(fh);
        NVStrings::destroy(strs);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];

    category_test(mode);
 }