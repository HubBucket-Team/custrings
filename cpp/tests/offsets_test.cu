
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include <numeric>
#include "../include/NVStrings.h"
#include "../include/NVCategory.h"
#include "../include/ipc_transfer.h"


//
// cd ../build
// nvcc -g -G -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/offsets_test.cu -L. -lNVStrings -lNVCategory -o offsets_test --linker-options -rpath,.:
//

int category_test( std::string& mode )
{
    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        FILE* fh = fopen("/tmp/cputx.bin","rb");

        size_t count_keys;
        size_t offsets_size;
        size_t strs_size;
        size_t count_strings;
        size_t values_size;
        
        fread(&count_keys,1,sizeof(size_t),fh);

        fread(&count_strings,1,sizeof(size_t),fh);

        fread(&values_size,1,sizeof(size_t),fh);

        int* values_ptr = (int*) malloc(values_size);
        fread(values_ptr,1,values_size,fh);

        fread(&offsets_size,1,sizeof(size_t),fh);

        int* offsets_ptr = (int*) malloc(offsets_size);
        fread(offsets_ptr,1,offsets_size,fh);

        fread(&strs_size,1,sizeof(size_t),fh);

        char* strs_ptr = (char*) malloc(strs_size);
        fread(strs_ptr,1,strs_size,fh);

        fclose(fh);

        cat = NVCategory::create_from_offsets(strs_ptr, count_keys, offsets_ptr);

        NVStrings* strs = cat->to_strings();

        printf("Received:\n");
        strs->print();

        printf("\nOriginal:\n");
        cat = cat->gather(values_ptr, count_strings, false);
        strs = cat->to_strings();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrs[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrs,5);
        cat = NVCategory::create_from_strings(*strs);
        printf("Original:\n");
        strs->print();

        strs = cat->get_keys();
        printf("\nSending keys:\n");
        strs->print();

        FILE* fh = fopen("/tmp/cputx.bin","wb");

        size_t count_keys = strs->size();
        fwrite((void*)&count_keys,1,sizeof(size_t),fh);

        size_t count_strings = cat->size();
        fwrite((void*)&count_strings,1,sizeof(size_t),fh);

        size_t offsets_size = (count_keys+1)*sizeof(int);
        int* offsets_ptr = (int*) malloc(offsets_size);

        int* lengths = (int*) malloc(count_keys*sizeof(int));
        strs->byte_count(lengths, false);

        size_t strs_size = std::accumulate(lengths, lengths+count_keys, 0);
        char* strs_ptr = (char*) malloc(strs_size);
        strs->create_offsets( strs_ptr, offsets_ptr, nullptr, false);

        size_t values_size = count_strings*sizeof(int);
        int* values_ptr = (int*) malloc(values_size);
        cat->get_values(values_ptr, false);

        fwrite((void*)&values_size,1,sizeof(size_t),fh);
        fwrite(values_ptr,1,values_size,fh);

        fwrite((void*)&offsets_size,1,sizeof(size_t),fh);
        fwrite(offsets_ptr,1,offsets_size,fh);

        fwrite((void*)&strs_size,1,sizeof(size_t),fh);
        fwrite(strs_ptr,1,strs_size,fh);

        fclose(fh);
        NVStrings::destroy(strs);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];

    category_test(mode);
 }