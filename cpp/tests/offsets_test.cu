
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include "../include/NVStrings.h"
#include "../include/NVCategory.h"
#include "../include/ipc_transfer.h"


//
// cd ../build
// nvcc -g -G -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/offsets_test.cu -L. -lNVStrings -lNVCategory -o offsets_test --linker-options -rpath,.:
//

int category_test( std::string& mode )
{
    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        FILE* fh = fopen("/tmp/cputx.bin","rb");

        size_t count;
        size_t offsets_size;
        size_t strs_size;
        
        fread(&count,1,sizeof(size_t),fh);

        fread(&offsets_size,1,sizeof(size_t),fh);

        int* offsets_ptr = (int*) malloc(offsets_size);
        fread(offsets_ptr,1,offsets_size,fh);

        fread(&strs_size,1,sizeof(size_t),fh);

        char* strs_ptr = (char*) malloc(strs_size);
        fread(strs_ptr,1,strs_size,fh);

        fclose(fh);

        cat = NVCategory::create_from_offsets(strs_ptr, count, offsets_ptr);

        NVStrings* strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrs[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrs,5);
        cat = NVCategory::create_from_strings(*strs);

        strs = cat->get_keys();
        strs->print();

        FILE* fh = fopen("/tmp/cputx.bin","wb");

        size_t count = strs->size();
        fwrite((void*)&count,1,sizeof(size_t),fh);

        size_t offsets_size = (strs->size()+1)*sizeof(int);
        int* offsets_ptr = (int*) malloc(offsets_size);

        size_t totalbytes = strs->total_bytes();
        char* strs_ptr = (char*) malloc(totalbytes);
        strs->create_offsets( strs_ptr, offsets_ptr, nullptr, false);

        size_t strs_size = offsets_ptr[strs->size()];

        fwrite((void*)&offsets_size,1,sizeof(size_t),fh);
        fwrite(offsets_ptr,1,offsets_size,fh);

        fwrite((void*)&strs_size,1,sizeof(size_t),fh);
        fwrite(strs_ptr,1,strs_size,fh);

        fclose(fh);
        NVStrings::destroy(strs);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];

    category_test(mode);
 }