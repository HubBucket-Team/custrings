
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <rmm/rmm.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include "../include/NVStrings.h"
#include "../include/NVCategory.h"
#include "../include/ipc_transfer.h"


//
// cd ../build
// nvcc -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/transfer_test.cu -I../../thirdparty/rmm/include -lrmm -L. -lNVStrings -lNVCategory -o transfer_test --linker-options -rpath,.:
// nvcc -g -G -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/transfer_test.cu -I../../thirdparty/rmm/include -lrmm -L. -lNVStrings -lNVCategory -o transfer_test --linker-options -rpath,.:
//

void print_transfer(nvcategory_transfer& ptr)
{
    printf("strs_size %d: ", ptr.strs_size);
    printf("vals_size %d: ", ptr.vals_size);
    printf("\n");
}

int category_test( std::string& mode )
{
    hipError_t cuda_error;

    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        nvcategory_transfer ptr;
        FILE* fh = fopen("/tmp/transfertx.bin","rb");

        fread(&ptr.base_address,1,sizeof(char*),fh);
        printf("base_address: %p\n", ptr.base_address);

        fread(&ptr.keys,1,sizeof(unsigned int),fh);
        printf("keys: %d\n", ptr.keys);

        fread(&ptr.strs_size,1,sizeof(size_t),fh);
        printf("strs_size: %d\n", ptr.strs_size);

        void* hstrs = (void*) malloc(ptr.strs_size);
        fread(hstrs,1,ptr.strs_size,fh);
        RMM_ALLOC(&ptr.strs, ptr.strs_size, 0);
        cuda_error = hipMemcpy(ptr.strs,hstrs,ptr.strs_size,hipMemcpyHostToDevice);
        if(cuda_error != hipSuccess) printf("Failed!\n");

        fread(&ptr.size,1,sizeof(size_t),fh);
        printf("size: %d\n", ptr.size);

        void* hmem = (void*) malloc(ptr.size);
        fread(hmem,1,ptr.size,fh);
        RMM_ALLOC(&ptr.mem, ptr.size, 0);
        cuda_error = hipMemcpy(ptr.mem,hmem,ptr.size,hipMemcpyHostToDevice);
        if(cuda_error != hipSuccess) printf("Failed!\n");

        fread(&ptr.count,1,sizeof(unsigned int),fh);
        printf("count: %d\n", ptr.count);

        fread(&ptr.vals_size,1,sizeof(size_t),fh);
        printf("vals_size: %d\n", ptr.vals_size);

        void* hvals = (void*) malloc(ptr.vals_size);
        fread(hvals,1,ptr.vals_size,fh);
        RMM_ALLOC(&ptr.vals, ptr.vals_size, 0);
        cuda_error = hipMemcpy(ptr.vals,hvals,ptr.vals_size,hipMemcpyHostToDevice);
        if(cuda_error != hipSuccess) printf("Failed!\n");

        fclose(fh);
        
        cat = NVCategory::create_from_transfer(ptr);
        printf("%p %p:%u %p:%u %p:%ld\n", ptr.base_address, ptr.strs, ptr.keys, ptr.vals, ptr.count, ptr.mem, ptr.size);
        NVStrings* strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrings[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrings,5);
        cat = NVCategory::create_from_strings(*strs);
        nvcategory_transfer ptr;
        cat->create_transfer(ptr);
        print_transfer(ptr);
        printf("%p %p:%u %p:%u %p:%ld\n", ptr.base_address, ptr.strs, ptr.keys, ptr.vals, ptr.count, ptr.mem, ptr.size);
        NVStrings::destroy(strs);
        strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);

        FILE* fh = fopen("/tmp/transfertx.bin","wb");

        printf("base_address: %p\n", ptr.base_address);
        fwrite((void*)&ptr.base_address,1,sizeof(char*),fh);
        fwrite((void*)&ptr.keys,1,sizeof(unsigned int),fh);

        fwrite((void*)&ptr.strs_size,1,sizeof(size_t),fh);

        void* hstrs = (void*) malloc(ptr.strs_size);
        cuda_error = hipMemcpy(hstrs,ptr.strs,ptr.strs_size,hipMemcpyDeviceToHost);
        if(cuda_error != hipSuccess) printf("Failed!\n");
        fwrite(&hstrs,1,ptr.strs_size,fh);

        fwrite((void*)&ptr.size,1,sizeof(size_t),fh);

        void* hmem = (void*) malloc(ptr.size);
        cuda_error = hipMemcpy(hmem,ptr.mem,ptr.size,hipMemcpyDeviceToHost);
        if(cuda_error != hipSuccess) printf("Failed!\n");
        fwrite(&hmem,1,ptr.size,fh);

        fwrite((void*)&ptr.count,1,sizeof(unsigned int),fh);
        fwrite((void*)&ptr.vals_size,1,sizeof(size_t),fh);

        void* hvals = (void*) malloc(ptr.vals_size);
        cuda_error = hipMemcpy(hvals,ptr.vals,ptr.vals_size,hipMemcpyDeviceToHost);
        if(cuda_error != hipSuccess) printf("Failed!\n");
        fwrite(&hvals,1,ptr.vals_size,fh);

        fclose(fh);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];
    //strings_test(mode);
    category_test(mode);
 }