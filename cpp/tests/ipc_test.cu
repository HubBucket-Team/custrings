
#include <stdio.h>
#include <string>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include "../include/NVStrings.h"
#include "../include/NVCategory.h"
#include "../include/ipc_transfer.h"


//
// cd ../build
// nvcc -w -std=c++11 --expt-extended-lambda -gencode arch=compute_70,code=sm_70 ../tests/ipc_test.cu -L. -lNVStrings -lNVCategory -o ipc_test --linker-options -rpath,.:
//

int category_test( std::string& mode )
{
    NVCategory* cat = 0;
    if( mode.compare("client")==0 )
    {
        nvcategory_ipc_transfer ipc;
        FILE* fh = fopen("ipctx.bin","rb");
        fread(&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        cat = NVCategory::create_from_ipc(ipc);
        //printf("%p %p:%u %p:%u %p:%ld\n", ipc.base_address, ipc.strs, ipc.keys, ipc.vals, ipc.count, ipc.mem, ipc.size);
        NVStrings* strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);
    }
    else
    {
        const char* hstrs[] = { "John", "Jane", "John", "Jane", "Bob" };
        NVStrings* strs = NVStrings::create_from_array(hstrs,5);
        cat = NVCategory::create_from_strings(*strs);
        nvcategory_ipc_transfer ipc;
        cat->create_ipc_transfer(ipc);
        //printf("%p %p:%u %p:%u %p:%ld\n", ipc.base_address, ipc.strs, ipc.keys, ipc.vals, ipc.count, ipc.mem, ipc.size);
        NVStrings::destroy(strs);
        strs = cat->get_keys();
        strs->print();
        NVStrings::destroy(strs);

        FILE* fh = fopen("ipctx.bin","wb");
        fwrite((void*)&ipc,1,sizeof(ipc),fh);
        fclose(fh);
        printf("Server ready. Press enter to terminate.\n");
        std::cin.ignore();
    }

    NVCategory::destroy(cat);
    return 0;
}

int main( int argc, const char** argv )
{
    if( argc < 2 )
    {
        printf("require parameter: 'server' or values for pointers\n");
        return 0;
    }
    std::string mode = argv[1];
    //strings_test(mode);
    category_test(mode);
 }